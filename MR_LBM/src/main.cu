
#include "main.cuh"
#include <iostream>
#include <chrono>
#include "saveData.cuh"

using namespace std;

std::map<unsigned int, unsigned int> cylinder_index;

int main()
{
	printf("BLOCK_NX: %d, BLOCK_NY: %d\n", BLOCK_NX, BLOCK_NY);

	folderSetup();

	// set cuda device
	checkCudaErrors(hipSetDevice(GPU_INDEX));

	// variable declaration
	dfloat* d_fMom;
	ghostInterfaceData ghostInterface;

	unsigned int* dNodeType;
	unsigned int* hNodeType;

	dfloat* h_fMom;

	dfloat* rho;

	dfloat* ux;
	dfloat* uy;

	dfloat* mxx;
	dfloat* mxy;
	dfloat* myy;

	/* ----------------- GRID AND THREADS DEFINITION FOR LBM ---------------- */
	dim3 threadBlock(BLOCK_NX, BLOCK_NY);
	dim3 gridBlock(NUM_BLOCK_X, NUM_BLOCK_Y);

	/* ------------------------- ALLOCATION FOR CPU ------------------------- */
	int step = 0;

	allocateHostMemory(&h_fMom, &rho, &ux, &uy, &mxx, &mxy, &myy);

	/* -------------- ALLOCATION FOR GPU ------------- */
	allocateDeviceMemory(&d_fMom, &dNodeType, &ghostInterface);

	// Setup Streams
	hipStream_t streamsLBM[1];
	checkCudaErrors(hipSetDevice(GPU_INDEX));
	checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
	checkCudaErrors(hipDeviceSynchronize());

	initializeDomain(ghostInterface, d_fMom, h_fMom, hNodeType, dNodeType, &step, gridBlock, threadBlock); 

	/* ------------------------------ TIMER EVENTS  ------------------------------ */
	checkCudaErrors(hipSetDevice(GPU_INDEX));
	hipEvent_t start, stop, start_step, stop_step;
	initializeCudaEvents(start, stop, start_step, stop_step);
	/* ------------------------------ LBM LOOP ------------------------------ */
	saveSimInfo(step, 0.0);

	/* --------------------------------------------------------------------- */
	/* ---------------------------- BEGIN LOOP ------------------------------ */
	/* --------------------------------------------------------------------- */
	for (step = INI_STEP; step < N_STEPS; step++)
	{
		gpuMomCollisionStream << <gridBlock, threadBlock >> > (d_fMom, dNodeType, ghostInterface, step);

		// swap interface pointers
		swapGhostInterfaces(ghostInterface);

		if (MACR_SAVE != 0 && step % MACR_SAVE == 0) {
			printf("\n----------------------------------- %d -----------------------------------\n", step);

			checkCudaErrors(hipDeviceSynchronize());
			checkCudaErrors(hipMemcpy(h_fMom, d_fMom, sizeof(dfloat) * NUMBER_LBM_NODES * NUMBER_MOMENTS, hipMemcpyDeviceToHost));
			saveMacr(h_fMom, rho, ux, uy, step);
		}
	}

	/* --------------------------------------------------------------------- */
	/* ------------------------------ END LOOP ------------------------------ */
	/* --------------------------------------------------------------------- */

	checkCudaErrors(hipDeviceSynchronize());

	// Calculate MLUPS

	dfloat MLUPS = recordElapsedTime(start_step, stop_step, step);
	printf("\n--------------------------- Last Time Step %06d ---------------------------\n", step);
	printf("MLUPS: %f\n", MLUPS);

	/* ------------------------------ POST ------------------------------ */
	checkCudaErrors(hipMemcpy(h_fMom, d_fMom, sizeof(dfloat) * NUMBER_LBM_NODES * NUMBER_MOMENTS, hipMemcpyDeviceToHost));
	// save info file
	saveSimInfo(step, MLUPS);

	/* ------------------------------ FREE ------------------------------ */
	hipFree(d_fMom);
	hipFree(dNodeType);
	hipFree(hNodeType);
	hipFree(hNodeType);
	hipFree(h_fMom);
	hipFree(rho);
	hipFree(ux);
	hipFree(uy);
	hipFree(mxx);
	hipFree(mxy);
	hipFree(myy);
	interfaceFree(ghostInterface);
	return 0;
}