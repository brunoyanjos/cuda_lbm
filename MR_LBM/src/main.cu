
#include "main.cuh"
#include <iostream>
#include <chrono>
#include "saveData.cuh"

using namespace std;

int main()
{
	printf("BLOCK_NX: %d, BLOCK_NY: %d\n", BLOCK_NX, BLOCK_NY);

	folderSetup();

	// set cuda device
	checkCudaErrors(hipSetDevice(GPU_INDEX));

	// variable declaration
	latticeNode *d_nodes;
	latticeNode *h_nodes;
	ghostInterfaceData ghostInterface;

	/* ----------------- GRID AND THREADS DEFINITION FOR LBM ---------------- */
	dim3 threadBlock(BLOCK_NX, BLOCK_NY);
	dim3 gridBlock(NUM_BLOCK_X, NUM_BLOCK_Y);

	/* ------------------------- ALLOCATION FOR CPU ------------------------- */
	int step = 0;

	allocateHostMemory(&h_nodes);

	/* -------------- ALLOCATION FOR GPU ------------- */
	allocateDeviceMemory(&d_nodes, &ghostInterface);

	// Setup Streams
	hipStream_t streamsLBM[1];
	checkCudaErrors(hipSetDevice(GPU_INDEX));
	checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
	checkCudaErrors(hipDeviceSynchronize());

	initializeDomain(ghostInterface, d_nodes, h_nodes, &step, gridBlock, threadBlock);

	/* ------------------------------ TIMER EVENTS  ------------------------------ */
	checkCudaErrors(hipSetDevice(GPU_INDEX));
	hipEvent_t start, stop, start_step, stop_step;
	initializeCudaEvents(start, stop, start_step, stop_step);
	/* ------------------------------ LBM LOOP ------------------------------ */
	saveSimInfo(step, 0.0);

	/* --------------------------------------------------------------------- */
	/* ---------------------------- BEGIN LOOP ------------------------------ */
	/* --------------------------------------------------------------------- */
	for (step = INI_STEP; step < N_STEPS; step++)
	{
		gpuMomCollisionStream<<<gridBlock, threadBlock>>>(d_nodes, ghostInterface, step);

		// swap interface pointers
		swapGhostInterfaces(ghostInterface);

		if (MACR_SAVE != 0 && step % MACR_SAVE == 0)
		{
			printf("\n----------------------------------- %d -----------------------------------\n", step);

			checkCudaErrors(hipDeviceSynchronize());
			checkCudaErrors(hipMemcpy(h_nodes, d_nodes, sizeof(latticeNode) * NUMBER_LBM_NODES, hipMemcpyDeviceToHost));

			kinetic_energy(h_nodes, step);
			//saveMacr(h_nodes, step);
		}
	}

	/* --------------------------------------------------------------------- */
	/* ------------------------------ END LOOP ------------------------------ */
	/* --------------------------------------------------------------------- */

	checkCudaErrors(hipDeviceSynchronize());

	// Calculate MLUPS

	dfloat MLUPS = recordElapsedTime(start_step, stop_step, step);
	printf("\n--------------------------- Last Time Step %06d ---------------------------\n", step);
	printf("MLUPS: %f\n", MLUPS);

	/* ------------------------------ POST ------------------------------ */
	checkCudaErrors(hipMemcpy(h_nodes, d_nodes, sizeof(latticeNode) * NUMBER_LBM_NODES, hipMemcpyDeviceToHost));
	// save info file
	saveSimInfo(step, MLUPS);
	velocity_profiles(h_nodes, step);

	/* ------------------------------ FREE ------------------------------ */
	hipFree(d_nodes);
	hipFree(h_nodes);
	interfaceFree(ghostInterface);
	return 0;
}