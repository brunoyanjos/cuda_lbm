
#include "main.cuh"
#include <iostream>
#include <chrono>
#include "saveData.cuh"

using namespace std;

std::map<unsigned int, unsigned int> cylinder_index;

int main()
{
	printf("BLOCK_NX: %d, BLOCK_NY: %d\n", BLOCK_NX, BLOCK_NY);

	folderSetup();

	// set cuda device
	checkCudaErrors(hipSetDevice(GPU_INDEX));

	// variable declaration
	dfloat *d_fMom;
	dfloat *d_fMom_old;
	ghostInterfaceData ghostInterface;
	cylinderProperties *d_cylinder_proporties;
	cylinderProperties *h_cylinder_proporties;

	unsigned int *dNodeType;
	unsigned int *hNodeType;

	dfloat D_Max;
	size_t countor_count;

	dfloat *h_fMom;

	dfloat *rho;

	dfloat *ux;
	dfloat *uy;

	dfloat *mxx;
	dfloat *mxy;
	dfloat *myy;

	/* ----------------- GRID AND THREADS DEFINITION FOR LBM ---------------- */
	dim3 threadBlock(BLOCK_NX, BLOCK_NY);
	dim3 gridBlock(NUM_BLOCK_X, NUM_BLOCK_Y);

	/* ------------------------- ALLOCATION FOR CPU ------------------------- */
	int step = 0;

	allocateHostMemory(&h_fMom, &rho, &ux, &uy, &mxx, &mxy, &myy);

	/* -------------- ALLOCATION FOR GPU ------------- */
	allocateDeviceMemory(&d_fMom, &dNodeType, &ghostInterface);

	hipMalloc(&d_fMom_old, MEM_SIZE_MOM);

	// Setup Streams
	hipStream_t streamsLBM[1];
	checkCudaErrors(hipSetDevice(GPU_INDEX));
	checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
	checkCudaErrors(hipDeviceSynchronize());

	initializeDomain(ghostInterface, d_fMom, h_fMom, hNodeType, dNodeType,
					 &step, gridBlock, threadBlock,
#ifdef CYLINDER
					 &D_Max, &h_cylinder_proporties,
					 d_cylinder_proporties, &countor_count
#endif
	);

	printf("count: %d, d_max:%f", countor_count, D_Max);

	const dfloat VISC = U_MAX * D_Max / RE;
	const dfloat TAU = 0.5 + 3.0 * VISC; // relaxation time

	const dfloat OMEGA = 1.0 / TAU;			   // (tau)^-1
	const dfloat OMEGAd2 = OMEGA / 2.0;		   // OMEGA/2
	const dfloat OMEGAd9 = OMEGA / 9.0;		   // OMEGA/9
	const dfloat T_OMEGA = 1.0 - OMEGA;		   // 1-OMEGA
	const dfloat TT_OMEGA = 1.0 - 0.5 * OMEGA; // 1.0 - OMEGA/2
	const dfloat OMEGA_P1 = 1.0 + OMEGA;	   // 1+ OMEGA
	const dfloat TT_OMEGA_T3 = TT_OMEGA * 3.0; // 3*(1-0.5*OMEGA)

	/* ------------------------------ TIMER EVENTS  ------------------------------ */
	checkCudaErrors(hipSetDevice(GPU_INDEX));
	hipEvent_t start, stop, start_step, stop_step;
	initializeCudaEvents(start, stop, start_step, stop_step);
	/* ------------------------------ LBM LOOP ------------------------------ */
	saveSimInfo(step, 0.0);

	/* --------------------------------------------------------------------- */
	/* ---------------------------- BEGIN LOOP ------------------------------ */
	/* --------------------------------------------------------------------- */
	for (step = INI_STEP; step < N_STEPS; step++)
	{
		#ifdef CYLINDER
		streamingAndMom << <gridBlock, threadBlock >> > (d_fMom, OMEGA, dNodeType, ghostInterface, d_cylinder_proporties, step);
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipMemcpy(d_fMom_old, d_fMom, sizeof(dfloat) * NUMBER_LBM_NODES * NUMBER_MOMENTS, hipMemcpyDeviceToDevice));

		boundaryAndCollision << <gridBlock, threadBlock >> > (d_fMom, d_fMom_old, OMEGA, dNodeType, ghostInterface, d_cylinder_proporties, step);
		#else
		gpuMomCollisionStream<<<gridBlock, threadBlock>>>(d_fMom, dNodeType, ghostInterface, step);
		#endif

		// swap interface pointers
		swapGhostInterfaces(ghostInterface);

		if (MACR_SAVE != 0 && step % MACR_SAVE == 0)
		{
			printf("\n----------------------------------- %d -----------------------------------\n", step);

			checkCudaErrors(hipDeviceSynchronize());
			checkCudaErrors(hipMemcpy(h_fMom, d_fMom, sizeof(dfloat) * NUMBER_LBM_NODES * NUMBER_MOMENTS, hipMemcpyDeviceToHost));
			saveMacr(h_fMom, rho, ux, uy, step);
		}
	}

	/* --------------------------------------------------------------------- */
	/* ------------------------------ END LOOP ------------------------------ */
	/* --------------------------------------------------------------------- */

	checkCudaErrors(hipDeviceSynchronize());

	// Calculate MLUPS

	dfloat MLUPS = recordElapsedTime(start_step, stop_step, step);
	printf("\n--------------------------- Last Time Step %06d ---------------------------\n", step);
	printf("MLUPS: %f\n", MLUPS);

	/* ------------------------------ POST ------------------------------ */
	checkCudaErrors(hipMemcpy(h_fMom, d_fMom, sizeof(dfloat) * NUMBER_LBM_NODES * NUMBER_MOMENTS, hipMemcpyDeviceToHost));
	// save info file
	saveSimInfo(step, MLUPS);

	/* ------------------------------ FREE ------------------------------ */
	hipFree(d_fMom);
	hipFree(dNodeType);
	hipFree(hNodeType);
	hipFree(hNodeType);
	hipFree(h_fMom);
	hipFree(rho);
	hipFree(ux);
	hipFree(uy);
	hipFree(mxx);
	hipFree(mxy);
	hipFree(myy);
	interfaceFree(ghostInterface);
	return 0;
}