#include "hip/hip_runtime.h"
#include "mlbm.cuh"
#include "globalStructs.h"
#include "globalFunctions.h"

__global__ void streamingAndMom(
    dfloat *fMom, dfloat OMEGA, unsigned int *dNodeType,
    ghostInterfaceData ghostInterface, cylinderProperties *cylinder_properties, unsigned int step)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x >= NX || y >= NY)
        return;

    dfloat pop[Q];
    dfloat pics2;
    dfloat multiplyTerm;
    __shared__ dfloat s_pop[BLOCK_LBM_SIZE * (Q - 1)];

    // Load moments from global memory

    // rho'
    unsigned int nodeType = dNodeType[idxScalarBlock(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y)];
    if (nodeType == 0b11111111)
        return;
    dfloat rhoVar = RHO_0 + fMom[idxMom(threadIdx.x, threadIdx.y, M_RHO_INDEX, blockIdx.x, blockIdx.y)];
    dfloat ux_t30 = fMom[idxMom(threadIdx.x, threadIdx.y, M_UX_INDEX, blockIdx.x, blockIdx.y)];
    dfloat uy_t30 = fMom[idxMom(threadIdx.x, threadIdx.y, M_UY_INDEX, blockIdx.x, blockIdx.y)];
    dfloat m_xx_t45 = fMom[idxMom(threadIdx.x, threadIdx.y, M_MXX_INDEX, blockIdx.x, blockIdx.y)];
    dfloat m_xy_t90 = fMom[idxMom(threadIdx.x, threadIdx.y, M_MXY_INDEX, blockIdx.x, blockIdx.y)];
    dfloat m_yy_t45 = fMom[idxMom(threadIdx.x, threadIdx.y, M_MYY_INDEX, blockIdx.x, blockIdx.y)];

    pop_reconstruction(rhoVar, ux_t30, uy_t30, m_xx_t45, m_xy_t90, m_yy_t45, pop);

    const unsigned short int xp1 = (threadIdx.x + 1 + BLOCK_NX) % BLOCK_NX;
    const unsigned short int xm1 = (threadIdx.x - 1 + BLOCK_NX) % BLOCK_NX;

    const unsigned short int yp1 = (threadIdx.y + 1 + BLOCK_NY) % BLOCK_NY;
    const unsigned short int ym1 = (threadIdx.y - 1 + BLOCK_NY) % BLOCK_NY;

    // save populations in shared memory
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 0)] = pop[1];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 1)] = pop[2];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 2)] = pop[3];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 3)] = pop[4];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 4)] = pop[5];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 5)] = pop[6];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 6)] = pop[7];
    s_pop[idxPopBlock(threadIdx.x, threadIdx.y, 7)] = pop[8];

    // sync threads of the block so all populations are saved
    __syncthreads();

    pop[1] = s_pop[idxPopBlock(xm1, threadIdx.y, 0)];
    pop[2] = s_pop[idxPopBlock(threadIdx.x, ym1, 1)];
    pop[3] = s_pop[idxPopBlock(xp1, threadIdx.y, 2)];
    pop[4] = s_pop[idxPopBlock(threadIdx.x, yp1, 3)];
    pop[5] = s_pop[idxPopBlock(xm1, ym1, 4)];
    pop[6] = s_pop[idxPopBlock(xp1, ym1, 5)];
    pop[7] = s_pop[idxPopBlock(xp1, yp1, 6)];
    pop[8] = s_pop[idxPopBlock(xm1, yp1, 7)];

    /* load pop from global in cover nodes */

    pop_load(ghostInterface, threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, pop);

    dfloat invRho;

    if (nodeType != BULK)
    {
        boundary_calculation(nodeType, &rhoVar, &ux_t30, &uy_t30, &m_xx_t45, &m_yy_t45, &m_xy_t90, pop, fMom, x, y, OMEGA);
    }
    else
    {
        rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8];
        invRho = 1 / rhoVar;

        ux_t30 = ((pop[1] + pop[5] + pop[8]) - (pop[3] + pop[6] + pop[7])) * invRho;
        uy_t30 = ((pop[2] + pop[5] + pop[6]) - (pop[4] + pop[7] + pop[8])) * invRho;

        m_xx_t45 = (pop[1] + pop[3] + pop[5] + pop[6] + pop[7] + pop[8]) * invRho - cs2;
        m_xy_t90 = ((pop[5] + pop[7]) - (pop[6] + pop[8])) * invRho;
        m_yy_t45 = (pop[2] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8]) * invRho - cs2;
    }

    // if (nodeType > 100 && step >= N_STEPS - FORCES_TIME && CALCULATE_FORCES) {
    // 	incoming_forces(nodeType, x, y, cylinder_properties, cylinder_index, pop);
    // }

    fMom[idxMom(threadIdx.x, threadIdx.y, M_RHO_INDEX, blockIdx.x, blockIdx.y)] = rhoVar - RHO_0;

    fMom[idxMom(threadIdx.x, threadIdx.y, M_UX_INDEX, blockIdx.x, blockIdx.y)] = ux_t30;
    fMom[idxMom(threadIdx.x, threadIdx.y, M_UY_INDEX, blockIdx.x, blockIdx.y)] = uy_t30;

    fMom[idxMom(threadIdx.x, threadIdx.y, M_MXX_INDEX, blockIdx.x, blockIdx.y)] = m_xx_t45;
    fMom[idxMom(threadIdx.x, threadIdx.y, M_MXY_INDEX, blockIdx.x, blockIdx.y)] = m_xy_t90;
    fMom[idxMom(threadIdx.x, threadIdx.y, M_MYY_INDEX, blockIdx.x, blockIdx.y)] = m_yy_t45;
}

__global__ void boundaryAndCollision(
    dfloat *fMom, dfloat *fMom_old, dfloat OMEGA, unsigned int *dNodeType,
    ghostInterfaceData ghostInterface, cylinderProperties *cylinder_properties, unsigned int step)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x >= NX || y >= NY)
        return;
    dfloat pop[Q];

    // Load moments from global memory

    // rho'
    unsigned int nodeType = dNodeType[idxScalarBlock(threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y)];
    if (nodeType == 0b11111111)
        return;
    dfloat rhoVar = RHO_0 + fMom[idxMom(threadIdx.x, threadIdx.y, M_RHO_INDEX, blockIdx.x, blockIdx.y)];
    dfloat ux_t30 = fMom[idxMom(threadIdx.x, threadIdx.y, M_UX_INDEX, blockIdx.x, blockIdx.y)];
    dfloat uy_t30 = fMom[idxMom(threadIdx.x, threadIdx.y, M_UY_INDEX, blockIdx.x, blockIdx.y)];
    dfloat m_xx_t45 = fMom[idxMom(threadIdx.x, threadIdx.y, M_MXX_INDEX, blockIdx.x, blockIdx.y)];
    dfloat m_xy_t90 = fMom[idxMom(threadIdx.x, threadIdx.y, M_MXY_INDEX, blockIdx.x, blockIdx.y)];
    dfloat m_yy_t45 = fMom[idxMom(threadIdx.x, threadIdx.y, M_MYY_INDEX, blockIdx.x, blockIdx.y)];

    // if (nodeType > 100) {
    // 	immersed_boundary_treatment(
    // 		nodeType,
    // 		&rhoVar, &ux_t30, &uy_t30, &m_xx_t45, &m_xy_t90, &m_yy_t45,
    // 		threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, x, y,
    // 		cylinder_properties,
    // 		fMom_old, OMEGA, step);
    // }

    ux_t30 = F_M_I_SCALE * ux_t30;
    uy_t30 = F_M_I_SCALE * uy_t30;

    m_xx_t45 = F_M_II_SCALE * (m_xx_t45);
    m_xy_t90 = F_M_IJ_SCALE * (m_xy_t90);
    m_yy_t45 = F_M_II_SCALE * (m_yy_t45);

    moment_collision(ux_t30, uy_t30, &m_xx_t45, &m_xy_t90, &m_yy_t45, OMEGA);

    pop_reconstruction(rhoVar, ux_t30, uy_t30, m_xx_t45, m_xy_t90, m_yy_t45, pop);

    // if (nodeType > 100 && step >= N_STEPS - FORCES_TIME && CALCULATE_FORCES) {
    // 	outgoing_forces(nodeType, x, y, cylinder_properties, cylinder_index, pop);
    // }

    fMom[idxMom(threadIdx.x, threadIdx.y, M_RHO_INDEX, blockIdx.x, blockIdx.y)] = rhoVar - RHO_0;

    fMom[idxMom(threadIdx.x, threadIdx.y, M_UX_INDEX, blockIdx.x, blockIdx.y)] = ux_t30;
    fMom[idxMom(threadIdx.x, threadIdx.y, M_UY_INDEX, blockIdx.x, blockIdx.y)] = uy_t30;

    fMom[idxMom(threadIdx.x, threadIdx.y, M_MXX_INDEX, blockIdx.x, blockIdx.y)] = m_xx_t45;
    fMom[idxMom(threadIdx.x, threadIdx.y, M_MXY_INDEX, blockIdx.x, blockIdx.y)] = m_xy_t90;
    fMom[idxMom(threadIdx.x, threadIdx.y, M_MYY_INDEX, blockIdx.x, blockIdx.y)] = m_yy_t45;

    pop_save(ghostInterface, threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, x, y, pop);
}